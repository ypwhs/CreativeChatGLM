#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// atomicAdd for double-precision floating-point numbers on hardware with
// compute capability < 6.0 from:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(
    double* address,
    double val
) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(
      address_as_ull,
      assumed,
      __double_as_longlong(val + __longlong_as_double(assumed))
    );

  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  		int* __restrict__ zeros,
    int batch,
    int vec_height, 	
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height, 	
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height, 	
    int height,
    int width,
    int zero_width,
    int groupsize
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height, 	
    int height,
    int width,
    int zero_width,
    int groupsize
);

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT2 =  16;
const int BLOCKHEIGHT3 =  24;
const int BLOCKHEIGHT4 =  32; 
const int BLOCKHEIGHT8 =  64;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

void vecquant2matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant2matmul_cuda", ([&] {
      VecQuant2MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant2MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 16;
  int k = 0;
  
  int z_w = w / 16; 
  int z_mod = (w % 16) * 2;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
	
    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod & 0x3) + 1);
	
    res += (scale * scalar_t((tmp >> 0) & 0x3) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 2) & 0x3) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 4) & 0x3) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 6) & 0x3) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 8) & 0x3) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 10) & 0x3) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 12) & 0x3) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 14) & 0x3) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp >> 16) & 0x3) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp >> 18) & 0x3) - zero) * blockvec[k + 9];
    res += (scale * scalar_t((tmp >> 20) & 0x3) - zero) * blockvec[k + 10];
    res += (scale * scalar_t((tmp >> 22) & 0x3) - zero) * blockvec[k + 11];
    res += (scale * scalar_t((tmp >> 24) & 0x3) - zero) * blockvec[k + 12];
    res += (scale * scalar_t((tmp >> 26) & 0x3) - zero) * blockvec[k + 13];
    res += (scale * scalar_t((tmp >> 28) & 0x3) - zero) * blockvec[k + 14];
    res += (scale * scalar_t((tmp >> 30) & 0x3) - zero) * blockvec[k + 15];
	
    i += width;
    k += 16;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = (h / 3) * 32;
  int k = 0;
  
  int z_w = (w / 32) * 3; // ((w / 256) * 24) / 3 
  int z_mod = w % 32;
  int z_bit;
  
  if (z_mod != 10){
    if (z_mod != 21){
      z_bit = z_mod;
      if (z_bit > 21){
        z_bit -= 22;
        z_bit *= 3;
        z_bit += 2;
        z_w += 2;
      } else if (z_bit > 10){
        z_bit -= 11;
        z_bit *= 3;
        z_bit += 1;
        z_w += 1;
      } else {
        z_bit *= 3;
      }
    } else {
      z_w += 1;
    }
  }
 
  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;
  unsigned int z_tmp;

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);
	
    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero;
    if (z_mod == 10) {
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 30) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 2) & 0x4);
      zero = scale * scalar_t((z_tmp) + 1);
    } else if (z_mod == 21){
      z_tmp = (as_unsigned(zeros[g * zero_width + z_w]) >> 31) | ((as_unsigned(zeros[g * zero_width + (z_w + 1)]) << 1) & 0x6);
      zero = scale * scalar_t((z_tmp) + 1);
    } else {
      zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_bit) & 0x7) + 1);
    }
	
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
	
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
	
    res += (scale * scalar_t((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
	
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
	
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
	
    i += width;
    k += 10;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant4matmul_cuda", ([&] {
      VecQuant4MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant4MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const       int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;

  int z_w = w / 8; 
  int z_mod = (w % 8) * 4;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);
	
    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1);
	
    res += (scale * scalar_t((tmp >> 0) & 0xF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 4) & 0xF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 8) & 0xF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 12) & 0xF) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp >> 16) & 0xF) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp >> 20) & 0xF) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp >> 24) & 0xF) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp >> 28) & 0xF) - zero) * blockvec[k + 7];
	
    i += width;
    k += 8;
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros,
  int groupsize
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1);
  int height = mat.size(0);
  int width = mat.size(1);
  int zero_width = zeros.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT8 - 1) / BLOCKHEIGHT8,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data<scalar_t>(), mat.data<int>(), mul.data<scalar_t>(),
        scales.data<scalar_t>(), zeros.data<int>(),
        batch, vec_height, height, width, zero_width, groupsize
      );
    })
  );
}

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  	int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize
) {
  int b = blockIdx.z;
  int h = BLOCKHEIGHT8 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * BLOCKWIDTH + threadIdx.x];
  __syncthreads();

  scalar_t res = 0;
  int i = width * h + w;
  int g_h = h * 4;
  int k = 0;
  
  int z_w = w / 4; 
  int z_mod = (w % 4) * 8;

  unsigned int tmp;

  while (k < BLOCKWIDTH) { 
    tmp = as_unsigned(mat[i]);
	
    int g = (g_h + k) / groupsize;
    scalar_t scale = scales[g * width + w];
    scalar_t zero = scale * scalar_t(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xFF) + 1);
	
    res += (scale * scalar_t((tmp >> 0) & 0xFF) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp >> 8) & 0xFF) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp >> 16) & 0xFF) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp >> 24) & 0xFF) - zero) * blockvec[k + 3];
	
    i += width;
    k += 4;
  }

  atomicAdd(&mul[b * width + w], res);
}
